#include "hip/hip_runtime.h"
#include "catch.hpp"

#include <pybind11/embed.h>
#include <pybind11/numpy.h>

#include "../include/iodata.h"
#include "../include/evaluate_density.cuh"
#include "../include/cuda_utils.cuh"
#include "../include/basis_to_gpu.cuh"
#include "../include/utils.h"

namespace py = pybind11;
using namespace py::literals;


TEST_CASE( "Test Electron Density Against gbasis", "[evaluate_electron_density_on_cubic]" ) {
  //py::initialize_interpreter();  // Open up the python interpretor for this test.
  {  // Need this so that the python object doesn't outline the interpretor.
  // Evaluate the electron density of this example.
  std::string fchk_file = GENERATE(
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_he.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_be.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_be_f_pure_orbital.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_kr.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_o.fchk",
//        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_c_g_pure_orbital.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_mg.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/E948_rwB97XD_def2SVP.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/h2o.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/ch4.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/qm9_000092_HF_cc-pVDZ.fchk"
        //"/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
  printf("IODATA OB %s \n", fchk_file.c_str());
  gbasis::IOData iodata = gbasis::get_molecular_basis_from_fchk(fchk_file);
  gbasis::UniformGrid grid =
      gbasis::get_grid_from_coordinates_charges(iodata.GetCoordAtoms(), iodata.GetCharges(), iodata.GetNatoms(), 0.0 );
  printf("DOne grid \n");
  double3 l_bnd = {grid.l_bnd[0], grid.l_bnd[1], grid.l_bnd[2]};
  std::array<int, 3> shape_arr = grid.calculate_shape({0.9, 0.9, 0.9});
  std::array<double, 9> axes_spacing = grid.multiply_axes_by_spacing(0.9, 0.9, 0.9);
  int3 shape = {shape_arr[0], shape_arr[1], shape_arr[2]};

  // Evaluate electron density on the cube
  printf("Electron density \n");
  gbasis::add_mol_basis_to_constant_memory_array(iodata.GetOrbitalBasis(), false, false);
  std::vector<double> result = gbasis::evaluate_electron_density_on_cubic(iodata, l_bnd, &axes_spacing[0], shape, false);
  printf("DOne \n");

  //Transfer result to pybind11 without copying
  pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast> py_result = gbasis::as_pyarray_from_vector(result);
  py::array_t<double> l_bnd_py(3, &l_bnd.x);
  py::array_t<double> axes_with_spacing_py(axes_spacing.size(), axes_spacing.data());
  py::array_t<int> shape_py(3, &shape.x);

  auto locals = py::dict(
      "true_result"_a=py_result, "fchk_path"_a=fchk_file, "l_bnd"_a=l_bnd_py, "axes"_a=axes_with_spacing_py, "shape"_a=shape_py
  );
  py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_density
from iodata import load_one
from gbasis.wrappers import from_iodata

iodata = load_one(fchk_path)
basis, type = from_iodata(iodata)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)

num_pts = 50
axes = np.reshape(axes, (3, 3)).T

random_indices = np.array([np.random.randint(0, shape[0], num_pts), np.random.randint(0, shape[1], num_pts), np.random.randint(0, shape[2], num_pts)]).T
global_index = random_indices[:, 0] * (shape[1] * shape[2]) + shape[2] * random_indices[:, 1] + random_indices[:, 2]
grid = l_bnd + random_indices.dot(axes)

density = evaluate_density(rdm, basis, grid, coord_type=type)

result = np.all(np.abs(density - true_result[global_index]) < 1e-8)
print(np.abs(density - true_result[global_index]))
assert result, "Electron density on GPU doesn't match gbasis."
    )", py::globals(), locals);

  if (!locals["result"].cast<bool>()) {
    REQUIRE(true);
  }
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
  //py::finalize_interpreter(); // Close up the python interpretor for this test.
}



TEST_CASE( "Test Electron Density Against gbasis on random grid", "[evaluate_electron_density_on_any_grid]" ) {
  //py::initialize_interpreter();  // Open up the python interpretor for this test.
  {  // Need this so that the python object doesn't outline the interpretor.
    // Evaluate the electron density of this example.
    std::string fchk_file = GENERATE(
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_01_H_N01_M2_ub3lyp_ccpvtz_g09.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_he.fchk"
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_be.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_be_f_pure_orbital.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_kr.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_o.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_c_g_pure_orbital.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/atom_mg.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/E948_rwB97XD_def2SVP.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/h2o.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/ch4.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/qm9_000092_HF_cc-pVDZ.fchk",
        "/home/ali-tehrani/SoftwareProjects/spec_database/tests/data/qm9_000104_PBE1PBE_pcS-3.fchk"
    );
    printf("IODATA OB %s \n", fchk_file.c_str());
    gbasis::IOData iodata = gbasis::get_molecular_basis_from_fchk(fchk_file);

    // Gemerate random grid.
    int numb_pts = 10000;
    std::vector<double> points(3 * numb_pts);
    std::random_device rnd_device;
    std::mt19937  merseene_engine {rnd_device()};
    std::uniform_real_distribution<double> dist {-5, 5};
    auto gen = [&dist, &merseene_engine](){return dist(merseene_engine);};
    std::generate(points.begin(), points.end(), gen);


    // Evaluate electron density on the cube
    printf("Electron density \n");
    gbasis::add_mol_basis_to_constant_memory_array(iodata.GetOrbitalBasis(), false, true);
    std::vector<double> result = gbasis::evaluate_electron_density_on_any_grid(iodata, points.data(), numb_pts);
    printf("DOne \n");

    //Transfer result to pybind11 without copying
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast> py_result =
        gbasis::as_pyarray_from_vector(result);
    pybind11::array_t<double, pybind11::array::c_style | pybind11::array::forcecast>
        py_points = gbasis::as_pyarray_from_vector(points);

    auto locals = py::dict(
        "true_result"_a=py_result, "fchk_path"_a=fchk_file, "points"_a = py_points, "numb_pts"_a = numb_pts
    );
    py::exec(R"(
import numpy as np
from gbasis.evals.density import evaluate_density
from iodata import load_one
from gbasis.wrappers import from_iodata

iodata = load_one(fchk_path)
basis, type = from_iodata(iodata)
rdm = (iodata.mo.coeffs * iodata.mo.occs).dot(iodata.mo.coeffs.T)
points = points.reshape((numb_pts, 3), order="F")
points = np.array(points, dtype=np.float64)

density = evaluate_density(rdm, basis, points, coord_type=type)

result = np.all(np.abs(density - true_result) < 1e-8)
print(np.abs(density - true_result))
assert result, "Electron density on GPU doesn't match gbasis."
    )", py::globals(), locals);

    if (!locals["result"].cast<bool>()) {
      REQUIRE(true);
    }
  } // Need this so that the python object doesn't outline the interpretor when we close it up.
  //py::finalize_interpreter(); // Close up the python interpretor for this test.
}
